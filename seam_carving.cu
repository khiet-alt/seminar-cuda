#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>

// ---------------------------------------- Utility function ----------------------------------------

#define CHECK(call)\
{\
    const hipError_t error = call;\
    if (error != hipSuccess)\
    {\
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);\
        fprintf(stderr, "code: %d, reason: %s\n", error,\
                hipGetErrorString(error));\
        exit(EXIT_FAILURE);\
    }\
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char * fileName, int &width, int &height, uchar3 * &pixels)
{
	FILE * f = fopen(fileName, "r");
	if (f == NULL)
	{
		printf("Cannot read %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	char type[3];
	fscanf(f, "%s", type);
	
	if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	fscanf(f, "%i", &width);
	fscanf(f, "%i", &height);
	
	int max_val;
	fscanf(f, "%i", &max_val);
	if (max_val > 255) // In this exercise, we assume 1 byte per value
	{
		fclose(f);
		printf("Cannot read %s\n", fileName); 
		exit(EXIT_FAILURE); 
	}

	pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
	for (int i = 0; i < width * height; i++)
		fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

	fclose(f);
}

void writePnm(uchar3 * pixels, int width, int height, char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	fprintf(f, "P3\n%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height; i++)
		fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);
	
	fclose(f);
}

void writeGrayscalePnm(uint8_t * pixels, int numChannels, int width, int height, 
		char * fileName)
{
	FILE * f = fopen(fileName, "w");
	if (f == NULL)
	{
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}	

	if (numChannels == 1)
		fprintf(f, "P2\n");
	else if (numChannels == 3)
		fprintf(f, "P3\n");
	else
	{
		fclose(f);
		printf("Cannot write %s\n", fileName);
		exit(EXIT_FAILURE);
	}

	fprintf(f, "%i\n%i\n255\n", width, height); 

	for (int i = 0; i < width * height * numChannels; i++)
		fprintf(f, "%hhu\n", pixels[i]);

	fclose(f);
}

float computeError(uchar3 * a1, uchar3 * a2, int n)
{
	float err = 0;
	for (int i = 0; i < n; i++)
	{
		err += abs((int)a1[i].x - (int)a2[i].x);
		err += abs((int)a1[i].y - (int)a2[i].y);
		err += abs((int)a1[i].z - (int)a2[i].z);
	}
	err /= (n * 3);
	return err;
}

void printError(uchar3 * deviceResult, uchar3 * hostResult, int width, int height)
{
	float err = computeError(deviceResult, hostResult, width * height);
	printf("Error: %f\n", err);
}

char * concatStr(const char * s1, const char * s2)
{
    char * result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
	hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n");
}

// ---------------------------------------- Sequential code -----------------------------------------

// Convert input image into grayscale image
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// uint8_t * outPixels: grayscale image
void convertToGrayscaleByHost(uchar3 * inPixels, int width, int height, uint8_t * outPixels)
{
    // gray = 0.299 * red + 0.587 * green + 0.114 * blue  
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            outPixels[i] = 0.299f * inPixels[i].x + 0.587f * inPixels[i].y + 0.114f * inPixels[i].z;
        }
    }
}

// Convert input image into energy matrix using Edge detection
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// uchar3 * energyMatrix: energy matrix
void edgeDetectionByHost(uint8_t * inPixels, int width, int height, uint8_t * energyMatrix)
{
	// X axis edge dectect
	int filterX[9] = {-1, 0, 1,
					  -2, 0, 2,
					  -1, 0, 1};
	// Y axis edge dectect
	int filterY[9] = {1, 2, 1,
					  0, 0, 0,
					 -1, -2, -1};
	int filterWidth = 3;

	for (int outPixelsR = 0; outPixelsR < height; outPixelsR++)
	{
		for (int outPixelsC = 0; outPixelsC < width; outPixelsC++)
		{
			float outPixelX = 0;
			float outPixelY = 0;
			for (int filterR = 0; filterR < filterWidth; filterR++)
			{
				for (int filterC = 0; filterC < filterWidth; filterC++)
				{
					float filterValX = filterX[filterR*filterWidth + filterC];
					float filterValY = filterY[filterR*filterWidth + filterC];

					int inPixelsR = outPixelsR - filterWidth/2 + filterR;
					int inPixelsC = outPixelsC - filterWidth/2 + filterC;
					inPixelsR = min(max(0, inPixelsR), height - 1);
					inPixelsC = min(max(0, inPixelsC), width - 1);
					uint8_t inPixel = inPixels[inPixelsR*width + inPixelsC];

					outPixelX += inPixel * filterValX;
					outPixelY += inPixel * filterValY;
				}
			}
			energyMatrix[outPixelsR*width + outPixelsC] = abs(outPixelX) + abs(outPixelY); 
		}
	}
}

void findSeamPathByHost1(uint8_t * inPixels, int width, int height, uint32_t * seamPath)
{
    uint32_t * path;
    path = (uint32_t *)malloc((height + 1) * sizeof(uint32_t));
    memset(path, 0, (height + 1) * sizeof(uint32_t));
	uint32_t minSum = 99999;

    for (int c = 0; c < width; c++) 
    {
        path[0] = c;
        path[height] = inPixels[c];
        int idx = 0;

        for (int r = 1; r < height; r++)
        {
            if (c == 0)
            {
                int mid = r * width + c;
                int right = r * width + (c + 1);
                
                idx = mid;
                if (inPixels[right] < inPixels[idx]) idx = right;
            }
            else if (c == width - 1)
            {
                int left = r * width + (c - 1);
                int mid = r * width + c;

                idx = left;
                if (inPixels[mid] < inPixels[idx]) idx = mid;             
            }
            else 
            {
                int left = r * width + (c - 1);
                int mid = r * width + c;
                int right = r * width + (c + 1);

                idx = left;
                if (inPixels[mid] < inPixels[idx]) idx = mid;
                if (inPixels[right] < inPixels[idx]) idx = right;
            }

            path[r] = idx;
            path[height] += inPixels[idx];
        }

        if (path[height] < minSum)
        {
            memcpy(seamPath, path, (height + 1) * sizeof(uint32_t));
			minSum = path[height];
        }
    }
	
	free(path);
}

void findSeamPathByHost2(uint8_t * inPixels, int width, int height, uint32_t * seamPath)
{
	uint32_t * minimalEnergy, * backtrack;
	backtrack = (uint32_t *)malloc(width * height * sizeof(uint32_t));
	minimalEnergy = (uint32_t *)malloc(width * height * sizeof(uint32_t));
	
	for (int c = 0; c < width; c++)
	{
		int idx = (height - 1) * width + c;
		minimalEnergy[idx] = inPixels[idx];
	}

    for (int r = height - 2; r >= 0; r--) 
    {
        int idx = 0;
        for (int c = 0; c < width; c++)
        {
            if (c == 0)
            {
                int mid = (r + 1) * width + c;
                int right = (r + 1) * width + (c + 1);
                
                idx = mid;
                if (minimalEnergy[right] < minimalEnergy[idx]) idx = right;
            }
            else if (c == width - 1)
            {
                int left = (r + 1) * width + (c - 1);
                int mid = (r + 1) * width + c;

                idx = left;
                if (minimalEnergy[mid] < minimalEnergy[idx]) idx = mid;             
            }
            else 
            {
                int left = (r + 1) * width + (c - 1);
                int mid = (r + 1) * width + c;
                int right = (r + 1) * width + (c + 1);

                idx = left;
                if (minimalEnergy[mid] < minimalEnergy[idx]) idx = mid;
                if (minimalEnergy[right] < minimalEnergy[idx]) idx = right;
            }
			
			int curIdx = r * width + c;
            minimalEnergy[curIdx] = inPixels[curIdx] + minimalEnergy[idx];
			backtrack[curIdx] = idx;
        }
    }
	
	uint32_t min = minimalEnergy[0];
	uint32_t minIdx = 0;
	for (int c = 1; c < width; c++) 
	{
		if (minimalEnergy[c] < min) 
		{
			min = minimalEnergy[c];
			minIdx = c;
		}
	}

	seamPath[0] = minIdx;
	int curIdx = minIdx;
	for (int r = 1; r < height; r++)
	{
		seamPath[r] = backtrack[curIdx];
		curIdx = backtrack[curIdx];
	}
	
	free(minimalEnergy);
	free(backtrack);
}

// Seam carving using host
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// int scale_width: image width after seam carving
// uchar3 * outPixels: image after seam carving
// int improvement: improvement version 1 if improvement = 1 
void seamCarvingByHost(uchar3 * inPixels, int width, int height, uchar3 * outPixels, 
        int scale_width, int improvement= 0)
{
    uchar3 * img = (uchar3 *)malloc(width * height * sizeof(uchar3));
    memcpy(img, inPixels, (width * height * sizeof(uchar3)));

    if (improvement == 0) printf("\nHost");
    else printf("\nHost improvement version 1");

	for (int i = 0; i < width - scale_width; i++)
    {
        int curWidth = width - i;
        uint8_t * grayScaleImg = (uint8_t *)malloc(curWidth * height * sizeof(uint8_t));
        uint8_t * edgeDetectImg = (uint8_t *)malloc(curWidth * height * sizeof(uint8_t));

		// TODO: Convert input image into grayscale image
        convertToGrayscaleByHost(img, curWidth, height, grayScaleImg);
		
        // TODO: Edge Detection
        edgeDetectionByHost(grayScaleImg, curWidth, height, edgeDetectImg);
        
        // TODO: Find Seam path and remove Seam path
        uint32_t * seamPath;
        uchar3 * temp;
        seamPath = (uint32_t *)malloc(height * sizeof(uint32_t));
        memset(seamPath, 0, height * sizeof(uint32_t));

        if (improvement == 0)
        {
            // TODO: Find Seam path using Greedy Algorithm
            findSeamPathByHost1(edgeDetectImg, curWidth, height, seamPath);
        } 
        else 
        {
            // TODO: Improvement version 1 -> Find Seam path using Dynamic Programming
			findSeamPathByHost2(edgeDetectImg, curWidth, height, seamPath);
        }
		
		temp = (uchar3 *)malloc((curWidth - 1) * height * sizeof(uchar3));

        int idx = 0;
        for (int r = 0; r < height; r++) 
        {
            for (int c = 0; c < curWidth; c++) 
            {
                int i = r * curWidth + c;
                if (i != seamPath[r])
                {
                    temp[idx] = img[i];
                    idx++;
                }
            }
        }

        img = (uchar3 *)realloc(img, (curWidth - 1) * height * sizeof(uchar3));
        memcpy(img, temp, (curWidth - 1) * height * sizeof(uchar3));
		
		free(grayScaleImg);
		free(edgeDetectImg);
        free(seamPath);
        free(temp);
    }

    memcpy(outPixels, img, scale_width * height * sizeof(uchar3));

    free(img);
}

// ----------------------------------------- Parallel code ------------------------------------------

// Convert input image into grayscale image kernel
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// uint8_t * outPixels: grayscale image
__global__ void convertToGrayscaleKernel(uchar3 * inPixels, int width, int height, 
		uint8_t * outPixels)
{
    // Reminder: gray = 0.299 * red + 0.587 * green + 0.114 * blue
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;	
	
    if (r < height && c < width)
    { 
        int i = r * width + c;
        outPixels[i] = 0.299f * inPixels[i].x + 0.587f * inPixels[i].y + 0.114f * inPixels[i].z;
    }	
}

// Convert input image into grayscale image
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// uint8_t * outPixels: grayscale image
void convertToGrayscaleByDevice(uchar3 * inPixels, int width, int height, uint8_t * outPixels, 
		dim3 blockSize=dim3(1))
{
	// TODO: Allocate device memories
	uchar3 * d_in;
	uint8_t * d_out;
	size_t nBytes = width * height * sizeof(uint8_t);
	CHECK(hipMalloc(&d_in, width * height * sizeof(uchar3)));
    CHECK(hipMalloc(&d_out, width * height * sizeof(uint8_t)));

	// TODO: Copy data to device memories
	CHECK(hipMemcpy(d_in, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

	// TODO: Set grid size and call kernel (remember to check kernel error)
	dim3 gridSize((width - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
	convertToGrayscaleKernel<<<gridSize, blockSize>>>(d_in, width, height, d_out);
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("ERROR: %s\n", hipGetErrorString(err));

	// TODO: Copy result from device memories
	CHECK(hipMemcpy(outPixels, d_out, nBytes, hipMemcpyDeviceToHost));

	// TODO: Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
}

// Seam carving using device
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// int scale_width: image width after seam carving
// uchar3 * outPixels: result
// int improvement: improvement version 2 -> 4 <=> improvement = 2 -> 4
// -> Improvement version 2: Parallel code
// -> Improvement version 3: Using SMEM for storing image matrix
// -> Improvement version 4: Using both SMEM and CMEM for storing kernel filter
void seamCarvingByDevice(uchar3 * inPixels, int width, int height, uchar3 * outPixels, 
        int scale_width, int improvement= 0)
{
	// TODO: Convert input image into grayscale image
	
	// TODO: Edge Detection
	
	// TODO: Find Seam path
	
	// TODO: Remove Seam path
}

// ----------------------------------------- Seam Carving -------------------------------------------

// Seam carving function for all case
// uchar3 * inPixels: input image
// int width: input image width
// int height: input image height
// int scale_width: image width after seam carving
// uchar3 * outPixels: result after seam carving	
// int improvement: improvement version	
void seamCarving(uchar3 * inPixels, int width, int height, uchar3 * outPixels, int scale_width, 
        bool useDevice= false, dim3 blockSize= dim3(1, 1), int improvement= 0)
{
	GpuTimer timer;
	timer.Start();
	
	if (useDevice == false)	// Use host
	{
		// TODO: Seam carving using host
        seamCarvingByHost(inPixels, width, height, outPixels, scale_width, improvement);
	}
	else // Use device
	{
		// TODO: Seam carving using device
	}
	
	timer.Stop();
    float time = timer.Elapsed();
	printf("\nRun time: %f ms\n", time);
}

// --------------------------------------------- Main -----------------------------------------------

int main(int argc, char ** argv)
{
	if (argc != 3 && argc != 4 && argc != 6)
	{
		printf("The number of arguments is invalid\n");
		return EXIT_FAILURE;
	}

	printDeviceInfo();

	// Read input image file
	int width, height;
	uchar3 * inPixels;
	readPnm(argv[1], width, height, inPixels);
	printf("\nInput image size (width x height): %i x %i\n", width, height);
    float scale_rate = 0.85;

    if (argc >= 4) 
    {
        scale_rate = atof(argv[3]);
    }
    int scale_width = width * scale_rate;
    printf("Output image size (width x height): %i x %i\n", scale_width, height);

	// Seam carving input image using host
	
	// No improvement
	uchar3 * outPixelsByHostNoImprovement = (uchar3 *)malloc(scale_width * height * sizeof(uchar3)); 
	seamCarving(inPixels, width, height, outPixelsByHostNoImprovement, scale_width);

	// Improvement version 1
	uchar3 * outPixelsByHostImprovement1 = (uchar3 *)malloc(scale_width * height * sizeof(uchar3)); 
	seamCarving(inPixels, width, height, outPixelsByHostImprovement1, scale_width, false, dim3(1, 1), 1);
	
    // Seam carving input image using device
    // dim3 blockSize(32, 32); // Default
	// if (argc == 6)
	// {
	// 	blockSize.x = atoi(argv[3]);
	// 	blockSize.y = atoi(argv[4]);
	// }	
	
	// Improvement version 2
	// uchar3 * outPixelsByDeviceImprovement2 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	// seamCarving(inPixels, width, height, outPixelsByDeviceImprovement2, scale_width, true, blockSize, 2);
	
	// Improvement version 3
	// uchar3 * outPixelsByDeviceImprovement3 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	// seamCarving(inPixels, width, height, outPixelsByDeviceImprovement3, scale_width, true, blockSize, 3);
	
	// Improvement version 4
	// uchar3 * outPixelsByDeviceImprovement4 = (uchar3 *)malloc(width * height * sizeof(uchar3));
	// seamCarving(inPixels, width, height, outPixelsByDeviceImprovement4, scale_width, true, blockSize, 4);

    // Write results to files
    char * outFileNameBase = strtok(argv[2], "."); // Get rid of extension
	writePnm(outPixelsByHostNoImprovement, scale_width, height, concatStr(outFileNameBase, "_host.pnm"));
	writePnm(outPixelsByHostImprovement1, scale_width, height, concatStr(outFileNameBase, "_host1.pnm"));
	// writePnm(outPixelsByDeviceImprovement2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
	// writePnm(outPixelsByDeviceImprovement3, width, height, concatStr(outFileNameBase, "_device3.pnm"));
	// writePnm(outPixelsByDeviceImprovement4, width, height, concatStr(outFileNameBase, "_device4.pnm"));
	

	// Free memories
	free(inPixels);
	free(outPixelsByHostNoImprovement);
	free(outPixelsByHostImprovement1);
	// free(outPixelsByDeviceImprovement2);
	// free(outPixelsByDeviceImprovement3);
	// free(outPixelsByDeviceImprovement4);
}